
#include <hip/hip_runtime.h>
__device__ int sign(double val) {
    if (val < 0) return -1;
    if (val > 0) return 1;
    return 0;
}

__device__ bool intersection(double p11x, double p11y, double p12x, double p12y,
                            double p21x, double p21y, double p22x, double p22y) {
    double cut1x = p12x - p11x;
    double cut1y = p12y - p11y;

    double prod1 = cut1x * (p21y - p11y) - cut1y * (p21x - p11x);
    double prod2 = cut1x * (p22y - p11y) - cut1y * (p22x - p11x);

    if (sign(prod1) == sign(prod2)) return false;
    double cut2x = p22x - p21x;
    double cut2y = p22y - p21y;

    prod1 = cut2x * (p11y - p21y) - cut2y * (p11x - p21x);
    prod2 = cut2x * (p12y - p21y) - cut2y * (p12x - p21x);
    
    return sign(prod1) != sign(prod2);
}


__device__ void reflect(double& rx, double& ry, double nx, double ny) {
    double mul = 2.0 * (rx * nx + ry * ny) / (nx * nx + ny * ny);
    rx -= nx * mul;
    ry -= ny * mul;
}

extern "C"
__global__ void move(
    double* particlePosX, double* particlePosY, 
    double* particleSpeedX, double* particleSpeedY,    
    double dt) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    particlePosX[i] += particleSpeedX[i] * dt;
    particlePosY[i] += particleSpeedY[i] * dt;
}

extern "C"
__global__ void update(
    double* particlePosX, double* particlePosY, 
    double* particleSpeedX, double* particleSpeedY,

    int wallCount,
    double* wallPos1X, double* wallPos1Y,
    double* wallPos2X, double* wallPos2Y,

    int acceleratorCount,
    double* acceleratorPos1X, double* acceleratorPos1Y,
    double* acceleratorPos2X, double* acceleratorPos2Y,
    double* acceleratorSpeedX, double* acceleratorSpeedY,
    
    double dt) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    double x = particlePosX[i];
    double y = particlePosY[i];

    for (int j = 0; j < acceleratorCount; j++) {
        double minX = min(acceleratorPos1X[j], acceleratorPos2X[j]);
        double maxX = max(acceleratorPos1X[j], acceleratorPos2X[j]);

        double minY = min(acceleratorPos1Y[j], acceleratorPos2Y[j]);
        double maxY = max(acceleratorPos1Y[j], acceleratorPos2Y[j]);

        if (x > minX && x < maxX && y > minY && y < maxY) {
            particleSpeedX[i] += acceleratorSpeedX[j] * dt;
            particleSpeedY[i] += acceleratorSpeedY[j] * dt;
        }
    }

    bool reflected;
    do {
        reflected = false;
        for (int j = 0; j < wallCount; j++) 
            if (intersection(x, y, x + particleSpeedX[i] * dt, y + particleSpeedY[i] * dt, wallPos1X[j], wallPos1Y[j], wallPos2X[j], wallPos2Y[j])) {
                reflect(particleSpeedX[i], particleSpeedY[i], wallPos1Y[j] - wallPos2Y[j], wallPos2X[j] - wallPos1X[j]);
                reflected = true;
                break;
            }
    } while (reflected);
}

extern "C"
__global__ void updateWithCollision(
    int particleCount,
    double* particlePosX, double* particlePosY, 
    double* particleSpeedX, double* particleSpeedY,
    double particleRadius, double particleCollisionMultiplier,

    int wallCount,
    double* wallPos1X, double* wallPos1Y,
    double* wallPos2X, double* wallPos2Y,

    int acceleratorCount,
    double* acceleratorPos1X, double* acceleratorPos1Y,
    double* acceleratorPos2X, double* acceleratorPos2Y,
    double* acceleratorSpeedX, double* acceleratorSpeedY,
    
    double dt) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    double x = particlePosX[i];
    double y = particlePosY[i];

    for (int j = 0; j < acceleratorCount; j++) {
        double minX = min(acceleratorPos1X[j], acceleratorPos2X[j]);
        double maxX = max(acceleratorPos1X[j], acceleratorPos2X[j]);

        double minY = min(acceleratorPos1Y[j], acceleratorPos2Y[j]);
        double maxY = max(acceleratorPos1Y[j], acceleratorPos2Y[j]);

        if (x > minX && x < maxX && y > minY && y < maxY) {
            particleSpeedX[i] += acceleratorSpeedX[j] * dt;
            particleSpeedY[i] += acceleratorSpeedY[j] * dt;
        }
    }

    for (int j = 0; j < particleCount; j++) {
        if (i != j) {
            double dx = x - particlePosX[j] + (particleSpeedX[i] - particleSpeedX[j]) * dt;
            double dy = y - particlePosY[j]  + (particleSpeedY[i] - particleSpeedY[j]) * dt;
            double dst = sqrt(dx * dx + dy * dy);
            if (dst < particleRadius) {
                dx /= dst;
                dy /= dst;
                double vel = sqrt(pow(particleSpeedX[j], 2) + pow(particleSpeedY[j], 2));
                double val = particleCollisionMultiplier * vel * (particleRadius - dst) / particleRadius * dt;
                particleSpeedX[i] += dx * val;
                particleSpeedY[i] += dy* val;
            }
        }
    }

    bool reflected;
    do {
        reflected = false;
        for (int j = 0; j < wallCount; j++) 
            if (intersection(x, y, x + particleSpeedX[i] * dt, y + particleSpeedY[i] * dt, wallPos1X[j], wallPos1Y[j], wallPos2X[j], wallPos2Y[j])) {
                reflect(particleSpeedX[i], particleSpeedY[i], wallPos1Y[j] - wallPos2Y[j], wallPos2X[j] - wallPos1X[j]);
                reflected = true;
                break;
            }
    } while (reflected);
}